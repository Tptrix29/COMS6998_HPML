#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 1000000  // Array size (adjustable)

// Kernel function for array addition
__global__ void add_arrays(float* a, float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;
    // Each thread processes multiple elements
    for (int i = 0; i < (size - 1) / totalThreads + 1; i++) {
        int index = i * totalThreads + idx;
        if (index < size) {
            c[index] = a[index] + b[index];
        }
    }
}

// Utility to check CUDA errors
void check_cuda_error(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Runs one scenario with unified memory
void run_scenario(int blocks, int threads_per_block, int size, int K) {
    std::cout << "Running with " << blocks << " blocks and "
              << threads_per_block << " threads per block, K = " << K << std::endl;

    size_t bytes = size * sizeof(float);

    // Allocate unified memory
    float *a, *b, *c;
    check_cuda_error(hipMallocManaged(&a, bytes), "hipMallocManaged a failed");
    check_cuda_error(hipMallocManaged(&b, bytes), "hipMallocManaged b failed");
    check_cuda_error(hipMallocManaged(&c, bytes), "hipMallocManaged c failed");

    // Initialize arrays directly (no need for separate host arrays)
    for (int i = 0; i < size; i++) {
        a[i] = 1;
        b[i] = 1;
    }

    // Warm up the GPU
    add_arrays<<<blocks, threads_per_block>>>(a, b, c, size);
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError(), "Kernel launch failed");

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start event
    hipEventRecord(start);

    // Launch kernel
    add_arrays<<<blocks, threads_per_block>>>(a, b, c, size);

    // Stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float duration = 0;
    hipEventElapsedTime(&duration, start, stop);
    std::cout << "Execution time: " << duration << " ms" << std::endl;

    // Cleanup events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy result back to host (optional for profiling, but keep for correctness)
    // Verify correctness with a few samples
    std::cout << "Sample results: c[0]=" << c[0] << ", c[size/2]=" << c[size/2] 
              << ", c[size-1]=" << c[size-1] << std::endl;
    // Check sum
    float sum = 0;
    for (int i = 0; i < size; i++) {
        sum += c[i];
    }
    std::cout << "Sum of c: " << sum << std::endl;

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

int main(int argc, char* argv[]) {
    // Check command-line arguments
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " K" << std::endl;
        std::cerr << "K is the size of arrays in millions" << std::endl;
        return 1;
    }

    int K = atoi(argv[1]);
    int array_size = K * N;

    // Scenario 1: 1 block, 1 thread
    run_scenario(1, 1, array_size, K);

    // Scenario 2: 1 block, 256 threads
    run_scenario(1, 256, array_size, K);

    // Scenario 3: Multiple blocks with 256 threads per block
    int threads = 256;
    int blocks = (array_size + threads - 1) / threads;
    run_scenario(blocks, threads, array_size, K);

    return 0;
}