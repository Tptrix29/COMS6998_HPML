#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 1000000  // Array size (adjustable)

// Kernel function for array addition
__global__ void add_arrays(float* a, float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

// Utility to check CUDA errors
void check_cuda_error(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Runs one scenario with unified memory
void run_scenario(int blocks, int threads_per_block, int size, int K) {
    std::cout << "Running with " << blocks << " blocks and "
              << threads_per_block << " threads per block, K = " << K << std::endl;

    size_t bytes = size * sizeof(float);

    // Allocate unified memory
    float *a, *b, *c;
    check_cuda_error(hipMallocManaged(&a, bytes), "hipMallocManaged a failed");
    check_cuda_error(hipMallocManaged(&b, bytes), "hipMallocManaged b failed");
    check_cuda_error(hipMallocManaged(&c, bytes), "hipMallocManaged c failed");

    // Initialize arrays directly (no need for separate host arrays)
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // Warm up the GPU
    add_arrays<<<blocks, threads_per_block>>>(a, b, c, size);
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError(), "Kernel launch failed");

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start event
    hipEventRecord(start);

    // Launch kernel K times
    for (int k = 0; k < K; k++) {
        add_arrays<<<blocks, threads_per_block>>>(a, b, c, size);
    }

    // Stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float duration = 0;
    hipEventElapsedTime(&duration, start, stop);
    std::cout << "Execution time: " << duration << " ms" << std::endl;

    // Cleanup events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

int main(int argc, char* argv[]) {
    int array_size = N;
    // Check command-line arguments
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " K" << std::endl;
        std::cerr << "K is the size of arrays in millions" << std::endl;
        return 1;
    }

    int K = atoi(argv[1]);

    // Scenario 1: 1 block, 1 thread
    run_scenario(1, 1, array_size, K);

    // Scenario 2: 1 block, 256 threads
    run_scenario(1, 256, array_size, K);

    // Scenario 3: Multiple blocks with 256 threads per block
    int threads = 256;
    int blocks = (array_size + threads - 1) / threads;
    run_scenario(blocks, threads, array_size, K);

    return 0;
}