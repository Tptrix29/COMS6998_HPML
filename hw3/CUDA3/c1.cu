#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iomanip>

#define H 1024
#define W 1024
#define C 3
#define P 1
#define FW 3
#define FH 3
#define K 64

// Kernel function to perform 2D convolution without tiling
__global__ void conv2d(double* input, double* filter, double* output) {
    // Calculate output position
    int w = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if the thread is within the output bounds
    if (w < W && h < H) {
        for (int k = 0; k < K; k++) {
            double sum = 0.0;

            for (int c = 0; c < C; c++) {
                for (int fh = 0; fh < FH; fh++) {
                    for (int fw = 0; fw < FW; fw++) {
                        // Calculate the corresponding input position (with padding adjustment)
                        int input_h = h + fh - P;
                        int input_w = w + fw - P;

                        // Check if the input position is within the valid input bounds
                        if (input_h >= 0 && input_h < H && input_w >= 0 && input_w < W) {
                            int input_idx = c * H * W + input_h * W + input_w;
                            int filter_idx = k * C * FH * FW + c * FH * FW + fh * FW + fw;

                            sum += input[input_idx] * filter[filter_idx];
                        }
                    }
                }
            }

            int output_idx = k * H * W + h * W + w;
            output[output_idx] = sum;
        }
    }
}

void check_cuda_error(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}


int main(int argc, char* argv[]) {
    double *h_input, *h_output, *h_filter;
    double *d_input, *d_output, *d_filter;
    size_t input_size = H * W * C * sizeof(double);
    size_t output_size = H * W * K * sizeof(double);
    size_t filter_size = K * C * FH * FW * sizeof(double);

    // Allocate host memory
    h_input = (double*)malloc(input_size);
    h_output = (double*)malloc(output_size);
    h_filter = (double*)malloc(filter_size);
    if (!h_input || !h_output || !h_filter) {
        std::cerr << "Failed to allocate host memory" << std::endl;
        return -1;
    }

    // Initialize input and filter arrays
    for (int c=0; c < C; ++c) {
        for (int h = 0; h < H; ++h) {
            for (int w = 0; w < W; ++w) {
                h_input[c * H * W + h * W + w] = c * (h + w);
            }
        }
    }
    for (int k = 0; k < K; ++k) {
        for (int c = 0; c < C; ++c) {
            for (int fh = 0; fh < FH; ++fh) {
                for (int fw = 0; fw < FW; ++fw) {
                    h_filter[k * C * FH * FW + c * FH * FW + fh * FW + fw] = (c + k) * (fh + fw);
                }
            }
        }
    }
    // Allocate device memory
    check_cuda_error(hipMalloc((void**)&d_input, input_size), "Failed to allocate device input memory");
    check_cuda_error(hipMalloc((void**)&d_output, output_size), "Failed to allocate device output memory");
    check_cuda_error(hipMalloc((void**)&d_filter, filter_size), "Failed to allocate device filter memory");
    // Copy input and filter arrays from host to device
    check_cuda_error(hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice), "Failed to copy input to device");
    check_cuda_error(hipMemcpy(d_filter, h_filter, filter_size, hipMemcpyHostToDevice), "Failed to copy filter to device");
    
    // Define grid and block sizes
    dim3 block(16, 16);
    dim3 grid((W + block.x - 1) / block.x, (H + block.y - 1) / block.y);

    // Warm up
    conv2d<<<grid, block>>>(d_input, d_filter, d_output);
    check_cuda_error(hipGetLastError(), "Kernel launch failed");
    check_cuda_error(hipDeviceSynchronize(), "Kernel execution failed");

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start event
    hipEventRecord(start);

    // Launch kernel
    conv2d<<<grid, block>>>(d_input, d_filter, d_output);
    check_cuda_error(hipGetLastError(), "Kernel launch failed");
    check_cuda_error(hipDeviceSynchronize(), "Kernel execution failed");

    // Stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float duration;
    hipEventElapsedTime(&duration, start, stop);
    std::cout << "Execution time: " << std::fixed << std::setprecision(3) << duration << " ms" << std::endl;
    
    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy output array from device to host
    check_cuda_error(hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost), "Failed to copy output to host");
    // Checksum
    double checksum = 0.0;
    for (int k = 0; k < K; ++k) {
        for (int h = 0; h < H; ++h) {
            for (int w = 0; w < W; ++w) {
                checksum += h_output[k * H * W + h * W + w];
            }
        }
    }
    std::cout << "Checksum: " << std::fixed << std::setprecision(0) << checksum << std::endl;
    
    // Free device memory
    check_cuda_error(hipFree(d_input), "Failed to free device input memory");
    check_cuda_error(hipFree(d_output), "Failed to free device output memory");
    check_cuda_error(hipFree(d_filter), "Failed to free device filter memory");
    
    // Free host memory
    free(h_input);
    free(h_output);
    free(h_filter);
    return 0;
}


