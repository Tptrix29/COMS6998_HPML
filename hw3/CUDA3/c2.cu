#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iomanip>

#define H 1024
#define W 1024
#define C 3
#define P 1
#define FW 3
#define FH 3
#define K 64

#define TILE_WIDTH 16

// Kernel function to perform 2D convolution without tiling
__global__ void conv2d(double* input, double* filter, double* output) {
    // For same convolution with P = 1, the shared memory tile size must include a 1-pixel border.
    const int tile_rows = TILE_WIDTH + 2 * P; // 16 + 2 = 18
    const int tile_cols = TILE_WIDTH + 2 * P; // 18

    // Allocate shared memory: one tile for each channel.
    __shared__ double sharedInput[C][tile_rows][tile_cols];

    // Compute the output pixel indices this thread is responsible for.
    int out_row = blockIdx.y * blockDim.y + threadIdx.y;
    int out_col = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.z;  // Each block in z-dimension handles one filter (kernel) index.

    // The top-left corner (in the global input) corresponding to the tile, without padding.
    int in_row_start = blockIdx.y * blockDim.y;
    int in_col_start = blockIdx.x * blockDim.x;

    // Each thread loads one or more elements of the shared tile.
    int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * blockDim.y;
    int numElements = tile_rows * tile_cols;

    // Load the tile (with halo) into shared memory.
    for (int idx = threadId; idx < numElements; idx += numThreads) {
        int i = idx / tile_cols;  // Row index in the shared tile.
        int j = idx % tile_cols;  // Column index in the shared tile.
        // Adjust index in global in put for padding offset
        int global_row = in_row_start - P + i;
        int global_col = in_col_start - P + j;
        for (int c = 0; c < C; c++) {
            if (global_row >= 0 && global_row < H && global_col >= 0 && global_col < W) {
                sharedInput[c][i][j] = input[c * H * W + global_row * W + global_col];
            } else {
                sharedInput[c][i][j] = 0.0;
            }
        }
    }

    __syncthreads();

    double sum = 0.0;
    // Compute convolution if the output pixel is within the bounds.
    if (out_row < H && out_col < W) {
        // Iterate over channels and the filter window.
        for (int c = 0; c < C; c++) {
            for (int fh = 0; fh < FH; fh++) {
                for (int fw = 0; fw < FW; fw++) {
                    // Use threadIdx plus the filter offset to index into shared memory.
                    double in_val = sharedInput[c][threadIdx.y + fh][threadIdx.x + fw];
                    // Filter is stored as [K][C][FH][FW].
                    double filter_val = filter[k * (C * FH * FW) +
                                               c * (FH * FW) +
                                               fh * FW +
                                               fw];
                    sum += in_val * filter_val;
                }
            }
        }
        output[k * (H * W) + out_row * W + out_col] = sum;
    }
}


void check_cuda_error(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}


int main(int argc, char* argv[]) {
    double *h_input, *h_output, *h_filter;
    double *d_input, *d_output, *d_filter;
    size_t input_size = H * W * C * sizeof(double);
    size_t output_size = H * W * K * sizeof(double);
    size_t filter_size = K * C * FH * FW * sizeof(double);

    // Allocate host memory
    h_input = (double*)malloc(input_size);
    h_output = (double*)malloc(output_size);
    h_filter = (double*)malloc(filter_size);
    if (!h_input || !h_output || !h_filter) {
        std::cerr << "Failed to allocate host memory" << std::endl;
        return -1;
    }

    // Initialize input and filter arrays
    for (int c = 0; c < C; ++c) {
        for (int h = 0; h < H; ++h) {
            for (int w = 0; w < W; ++w) {
                h_input[c * H * W + h * W + w] = c * (h + w);
            }
        }
    }
    for (int k = 0; k < K; ++k) {
        for (int c = 0; c < C; ++c) {
            for (int fh = 0; fh < FH; ++fh) {
                for (int fw = 0; fw < FW; ++fw) {
                    h_filter[k * C * FH * FW + c * FH * FW + fh * FW + fw] = (c + k) * (fh + fw);
                }
            }
        }
    }
    // Allocate device memory
    check_cuda_error(hipMalloc((void**)&d_input, input_size), "Failed to allocate device input memory");
    check_cuda_error(hipMalloc((void**)&d_output, output_size), "Failed to allocate device output memory");
    check_cuda_error(hipMalloc((void**)&d_filter, filter_size), "Failed to allocate device filter memory");
    // Copy input and filter arrays from host to device
    check_cuda_error(hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice), "Failed to copy input to device");
    check_cuda_error(hipMemcpy(d_filter, h_filter, filter_size, hipMemcpyHostToDevice), "Failed to copy filter to device");
    
    // Define grid and block sizes
    dim3 block(16, 16, 1);
    dim3 grid((W + block.x - 1) / block.x, (H + block.y - 1) / block.y, K);

    // Warm up
    conv2d<<<grid, block>>>(d_input, d_filter, d_output);
    check_cuda_error(hipGetLastError(), "Kernel launch failed");
    check_cuda_error(hipDeviceSynchronize(), "Kernel execution failed");
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start event
    hipEventRecord(start);

    // Launch kernel
    conv2d<<<grid, block>>>(d_input, d_filter, d_output);
    check_cuda_error(hipGetLastError(), "Kernel launch failed");
    check_cuda_error(hipDeviceSynchronize(), "Kernel execution failed");

    // Stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float duration = 0;
    hipEventElapsedTime(&duration, start, stop);
    std::cout << "Execution time: " << std::fixed << std::setprecision(3) << duration << " ms" << std::endl;

    // Cleanup events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy output array from device to host
    check_cuda_error(hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost), "Failed to copy output to host");
    // Checksum
    double checksum = 0.0;
    for (int k = 0; k < K; ++k) {
        for (int h = 0; h < H; ++h) {
            for (int w = 0; w < W; ++w) {
                checksum += h_output[k * H * W + h * W + w];
            }
        }
    }
    std::cout << "Checksum: " << std::fixed << std::setprecision(0) << checksum << std::endl;
    // Free device memory
    check_cuda_error(hipFree(d_input), "Failed to free device input memory");
    check_cuda_error(hipFree(d_output), "Failed to free device output memory");
    check_cuda_error(hipFree(d_filter), "Failed to free device filter memory");
    // Free host memory
    free(h_input);
    free(h_output);
    free(h_filter);
    return 0;
}


